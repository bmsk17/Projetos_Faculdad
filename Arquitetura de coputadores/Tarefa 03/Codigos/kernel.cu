#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

const int matrixSizes[] = { 100, 200, 300, 400, 500 };  // Tamanhos das matrizes

__global__ void matrixMultiplicationKernel(const int* A, const int* B, int* result, int matrixSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < matrixSize && col < matrixSize) {
        int value = 0;
        for (int k = 0; k < matrixSize; ++k) {
            value += A[row * matrixSize + k] * B[k * matrixSize + col];
        }
        result[row * matrixSize + col] = value;
    }
}

void matrixMultiplicationParallel(const int matrixSize, const int numThreads) {
    // Matrizes na CPU
    int* hostMatrixA = new int[matrixSize * matrixSize];
    int* hostMatrixB = new int[matrixSize * matrixSize];
    int* hostResultMatrix = new int[matrixSize * matrixSize];

    // Preencher as matrizes com dados (para este exemplo, você pode ajustar conforme necessário)
    for (int i = 0; i < matrixSize * matrixSize; ++i) {
        hostMatrixA[i] = i;
        hostMatrixB[i] = i;
    }

    // Matrizes na GPU
    int* deviceMatrixA, * deviceMatrixB, * deviceResultMatrix;
    hipMalloc((void**)&deviceMatrixA, matrixSize * matrixSize * sizeof(int));
    hipMalloc((void**)&deviceMatrixB, matrixSize * matrixSize * sizeof(int));
    hipMalloc((void**)&deviceResultMatrix, matrixSize * matrixSize * sizeof(int));

    // Transferir dados para a GPU
    hipMemcpy(deviceMatrixA, hostMatrixA, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, hostMatrixB, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);

    // Configurar grade e blocos
    dim3 blockDim(numThreads, numThreads);
    dim3 gridDim((matrixSize + blockDim.x - 1) / blockDim.x, (matrixSize + blockDim.y - 1) / blockDim.y);

    // Criar eventos CUDA para medir o tempo de execução
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Registrar o início do tempo de execução
    hipEventRecord(start);

    // Executar o kernel na GPU para multiplicar as matrizes
    matrixMultiplicationKernel << <gridDim, blockDim >> > (deviceMatrixA, deviceMatrixB, deviceResultMatrix, matrixSize);
    hipDeviceSynchronize();  // Aguardar a conclusão do kernel

    // Registrar o fim do tempo de execução
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcular o tempo de execução em milissegundos
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Imprimir o resultado formatado
    std::cout << "Threads: " << numThreads << " , tempo de execucao: " << milliseconds << " ms" << std::endl;

    // Transferir o resultado de volta para a CPU (opcional)
    // hipMemcpy(hostResultMatrix, deviceResultMatrix, matrixSize * matrixSize * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memória
    delete[] hostMatrixA;
    delete[] hostMatrixB;
    delete[] hostResultMatrix;
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceResultMatrix);
}

int main() {
    const int numThreads[] = { 1, 2, 4, 8, 16, 32 };  // Diferentes números de threads por bloco
    for (int i = 0; i < sizeof(matrixSizes) / sizeof(matrixSizes[0]); ++i) {
        std::cout << "Tamanho da Matriz " << matrixSizes[i] << " x " << matrixSizes[i] << ":" << std::endl;
        for (int j = 0; j < sizeof(numThreads) / sizeof(numThreads[0]); ++j) {
            matrixMultiplicationParallel(matrixSizes[i], numThreads[j]);
        }
        std::cout << "--------------------------------------------------------" << std::endl;
    }

    return 0;
}
